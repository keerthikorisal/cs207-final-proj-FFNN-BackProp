
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>



#define TILE_SIZE 8

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
	__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
	__shared__ float ds_B[TILE_SIZE][TILE_SIZE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
        int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * blockDim.y + ty;
	int col = bx * blockDim.x + tx;
	float pvalue = 0;

	for ( int p = 0; p < (k - 1)/TILE_SIZE+1; p++){

		if((p*TILE_SIZE + tx < k) && (row < m)){
			ds_A[ty][tx] = A[row * k + p * TILE_SIZE+tx];
		}
		else ds_A[ty][tx] = 0;
		
		if((p*TILE_SIZE + ty < k) && (col < n)){
			ds_B[ty][tx] = B[(p*TILE_SIZE+ty)*n + col];
		}
		else ds_B[ty][tx] = 0;
		__syncthreads();

		if(row < m && col < n){
			for (int i = 0; i <  TILE_SIZE; ++i){
				pvalue += ds_A[ty][i] * ds_B[i][tx];
			}
		}
		__syncthreads();

		if(row < m && col < n){
			C[row * n + col] = pvalue;
		}

	}
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
	dim3 dimGrid((n-1)/BLOCK_SIZE + 1, (m-1) / BLOCK_SIZE +1, 1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
 
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	mysgemm <<< dimGrid, dimBlock >>> (m, n, k, A, B, C);	
    /*************************************************************************/


}

__device__ __forceinline__ float sigmoid (float a){
	return 1.0 / (1.0 + exp (-a));
}

__global__ void sigmoid_kernel (const float *C, float *Csig, int len){
	int stride = gridDim.x * blockDim.x;
    	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = tid; i < len; i += stride){
		Csig[i] = sigmoid (C[i]);
	}

}

void basicSigmoid(const float *C, float  *Csig, int len){
	dim3 dimBlock(256);
	int threadBlocks = (len + (dimBlock.x - 1)) / dimBlock.x;
	if (threadBlocks > 65520) threadBlocks = 65520;
	dim3 dimGrid(threadBlocks);

	sigmoid_kernel<<<dimGrid,dimBlock>>>(C, Csig, len);

}


__global__ void back_prop_kernel(const float *Target, float *OW, float *OUT, float *C, int backprop_len){

	int stride = gridDim.x * blockDim.x;
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float *new_w;
	int m, n, k;
        m = n = k = 2;
       // basicSgemm(m, n, k, OW, C, OUT);

	if(backprop_len < 0){
		int m, n, k;
		m = n = k = 2;
		//basicSgemm(m, n, k, OW, C, OUT);
		for (int i = tid; i < backprop_len; i += stride){
                	new_w[i] = 1 ;
       		}
	}

}

void basicBackProp(const float *Target, float *OW, float *OUT, float *C, int backprop_len){
	dim3 dimBlock(256);
	int threadBlocks = (backprop_len + (dimBlock.x - 1)) / dimBlock.x;
	if (threadBlocks > 65520) threadBlocks = 65520;
        dim3 dimGrid(threadBlocks);

        back_prop_kernel<<<dimGrid,dimBlock>>>(Target, OW, OUT, C, backprop_len);

}
