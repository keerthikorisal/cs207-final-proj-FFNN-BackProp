
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>



#define TILE_SIZE 8

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
	__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
	__shared__ float ds_B[TILE_SIZE][TILE_SIZE];

	int bx = blockIdx.x;
	int by = blockIdx.y;
        int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * blockDim.y + ty;
	int col = bx * blockDim.x + tx;
	float pvalue = 0;

	for ( int p = 0; p < (k - 1)/TILE_SIZE+1; p++){

		if((p*TILE_SIZE + tx < k) && (row < m)){
			ds_A[ty][tx] = A[row * k + p * TILE_SIZE+tx];
		}
		else ds_A[ty][tx] = 0;
		
		if((p*TILE_SIZE + ty < k) && (col < n)){
			ds_B[ty][tx] = B[(p*TILE_SIZE+ty)*n + col];
		}
		else ds_B[ty][tx] = 0;
		__syncthreads();

		if(row < m && col < n){
			for (int i = 0; i <  TILE_SIZE; ++i){
				pvalue += ds_A[ty][i] * ds_B[i][tx];
			}
		}
		__syncthreads();

		if(row < m && col < n){
			C[row * n + col] = pvalue;
		}

	}
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
	dim3 dimGrid((n-1)/BLOCK_SIZE + 1, (m-1) / BLOCK_SIZE +1, 1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
 
    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	mysgemm <<< dimGrid, dimBlock >>> (m, n, k, A, B, C);	
    /*************************************************************************/


}

__device__ __forceinline__ float sigmoid (float a){
	return 1.0 / (1.0 + exp (-a));
}

__global__ void sigmoid_kernel (const float *C, float *Csig, int len){
	int stride = gridDim.x * blockDim.x;
    	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = tid; i < len; i += stride){
		Csig[i] = sigmoid (C[i]);
	}

}

void basicSigmoid(const float *C, float  *Csig, int len){
	dim3 dimBlock(256);
	int threadBlocks = (len + (dimBlock.x - 1)) / dimBlock.x;
	if (threadBlocks > 65520) threadBlocks = 65520;
	dim3 dimGrid(threadBlocks);

	sigmoid_kernel<<<dimGrid,dimBlock>>>(C, Csig, len);

}


__global__ void mysub(int m, int n, int k, const float *A, const float *B, float* C){

	// INSERT KERNEL CODE HERE
        //__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
        //__shared__ float ds_B[TILE_SIZE][TILE_SIZE];

       /* int bx = blockIdx.x;
        int by = blockIdx.y;
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        int row = by * blockDim.y + ty;
        int col = bx * blockDim.x + tx;
        //float pvalue = 0;

       * for (int p = 0; p < (k - 1)/TILE_SIZE+1; p++){

                if(row < m && col < n){
                        for (int i = 0; i <  TILE_SIZE; ++i){
                                pvalue = ds_A[ty][i] - ds_B[ty][i];
                        }
                }
                __syncthreads();

                if(row < m && col < n){
                        C[row * n + col] = pvalue;
                }

        }/
	int id = gridDim.x * by + bx;
	C[id] = A[id] - B[id];*/
	int colID = threadIdx.x + blockIdx.x * blockDim.x;	// Row address

	if(colID < n) {
		for(int i = 0; i<n; i++){
			//elemID = colID + rowID * WIDTH; 
			C[colID + i*n] = A[colID + i*n] + B[colID + i*n];
		}
	}	

}

void basicSub(int m, int n, int k, const float *A, const float *B, float* C){
	// Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    /*************************************************************************/
    //INSERT CODE HERE
        dim3 dimGrid((n+1)/BLOCK_SIZE + 1, (m + 1) / BLOCK_SIZE, 1);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
        mysub <<< dimGrid, dimBlock >>> (m, n, k, A, B, C);
    /*************************************************************************/


}

__global__ void mytrans(int m, int n, const float *OW_d, float *OW_t_d){
	__shared__ float tile[TILE_SIZE][TILE_SIZE];

	int x = blockIdx.x * TILE_SIZE + threadIdx.x;
	int y = blockIdx.y * TILE_SIZE + threadIdx.y;
 	if((x < m) && (y < n))
	{
		unsigned int index_in = y * m + x;
		tile[threadIdx.y][threadIdx.x] = OW_d[index_in];
	}
	__syncthreads();

	x = blockIdx.y * TILE_SIZE + threadIdx.x;
	y = blockIdx.x * TILE_SIZE + threadIdx.y;
	if((x < n) && (y < m))
	{
		unsigned int index_out = y * n + x;
		OW_t_d[index_out] = tile[threadIdx.x][threadIdx.y];
	}

}

void basicTrans(int m, int n, const float *OW_d, float *OW_t_d){
	
	// Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    /*************************************************************************/
    //INSERT CODE HERE
        dim3 dimGrid(m / BLOCK_SIZE, n / BLOCK_SIZE, 1);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
        mytrans <<< dimGrid, dimBlock >>> (m, n, OW_d, OW_t_d);
    /*************************************************************************/
			

}
