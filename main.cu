#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"
#include <iostream>

void feed_forward(unsigned matArow, unsigned matBcol, unsigned matBrow, int len, int OUT_len, float *OW_d, float *C_d, float *Csig_d, float *OUT_d, float *A_d, float *B_d, hipError_t cuda_ret){
    	cuda_ret = hipDeviceSynchronize();
	basicSgemm(matArow, matBcol, matBrow, A_d, B_d, C_d);
    	cuda_ret = hipDeviceSynchronize();
    	basicSigmoid(C_d, Csig_d, len);
    	cuda_ret = hipDeviceSynchronize();
    	basicSgemm(matBcol, matBcol, matArow, OW_d, Csig_d, OUT_d);
    	cuda_ret = hipDeviceSynchronize();
    	basicSigmoid(OUT_d, OUT_d,  OUT_len);
    	cuda_ret = hipDeviceSynchronize();
}

void back_prop_output(const float *OW_d, float *C_d, float *OW_new_d, float *update_weight_d, float *OW_t_d,  hipError_t cuda_ret){
	unsigned matArow, matBrow, matBcol;
	matArow = 1; matBrow = 5; matBcol = 1;
	cuda_ret = hipDeviceSynchronize();
	basicSgemm(matBrow, matBcol, matArow, C_d, update_weight_d, OW_new_d);
	cuda_ret = hipDeviceSynchronize();
	cuda_ret = hipDeviceSynchronize();
	basicSub(matBcol, matBrow, matArow, OW_d, OW_new_d, OW_new_d);
	cuda_ret = hipDeviceSynchronize();
}

void back_prop_hidden(float *B_d, float *update_weight_d, float *A_d, float *OW_d, float *OW_t_d, float *B_new_d, float *B_temp_d, hipError_t cuda_ret){
	unsigned matArow, matBrow, matBcol;
        matArow = 4; matBrow = 5; matBcol = 1;
        cuda_ret = hipDeviceSynchronize();
	basicSgemm(matArow, matBcol, matBcol, B_d, update_weight_d, B_temp_d);
	cuda_ret = hipDeviceSynchronize();
	basicSgemm(matArow, matBrow, matBcol, B_temp_d, OW_d,  B_new_d);
        cuda_ret = hipDeviceSynchronize();
	basicSub(matBrow, matArow, matBcol, A_d, B_new_d, B_new_d);
	cuda_ret = hipDeviceSynchronize();
}
 

int main (int argc, char *argv[])
{

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);
    
    float *A_h, *B_h, *C_h, *Csig_h, *OW_h, *OUT_h, *OW_new_h, *update_weight_h, *OW_t_h, *B_new_h, *B_temp_h;
    float *A_d, *B_d, *C_d, *Csig_d, *OW_d, *OUT_d, *OW_new_d, *update_weight_d, *OW_t_d, *B_new_d, *B_temp_d;
    size_t A_sz, B_sz, C_sz, Csig_sz, OW_sz, OUT_sz, OW_new_sz, update_sz, OW_t_sz, B_new_sz, B_temp_sz;
    int len, OUT_len;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./sgemm-tiled                # All matrices are 1000 x 1000"
      "\n    Usage: ./sgemm-tiled <m>            # All matrices are m x m"
      "\n    Usage: ./sgemm-tiled <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
      "\n");
        exit(0);
    }
    
    //Matrix sizes
    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;
    Csig_sz = C_sz;
    OW_sz = matArow*matBcol;
    OUT_sz = matBcol;
    len = C_sz;
    OUT_len = OUT_sz;
    OW_new_sz = OW_sz;
    update_sz = OUT_sz;
    OW_t_sz = A_sz;
    B_new_sz = A_sz;
    B_temp_sz = B_sz;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    Csig_h = (float*) malloc( sizeof(float)*Csig_sz);
    for (unsigned int i = 0; i < Csig_sz; i++) { Csig_h[i] = (rand()%100)/100.00; }

    OW_h = (float*) malloc(sizeof(float)*OW_sz);
    for(unsigned int i = 0; i < OW_sz; i++) { OW_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );
    OUT_h = (float*) malloc(sizeof(float)*OUT_sz);
    OW_new_h = (float*) malloc(sizeof(float)*OW_new_sz);
    update_weight_h = (float*) malloc(sizeof(float)*update_sz);	
    OW_t_h = (float*) malloc(sizeof(float)*OW_t_sz);	
    B_new_h = (float*) malloc( sizeof(float)*B_new_sz);
    B_temp_h = (float*) malloc( sizeof(float)*B_temp_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matArow, matAcol,
        matBrow, matBcol, matArow, matBcol);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    hipMalloc((void**) &A_d, sizeof(float)*A_sz);
    hipMalloc((void**) &B_d, sizeof(float)*B_sz);
    hipMalloc((void**) &C_d, sizeof(float)*C_sz);
    hipMalloc((void**) &OW_d, sizeof(float)*OW_sz);
    hipMalloc((void**) &OUT_d, sizeof(float)*OUT_sz);
    hipMalloc((void**) &Csig_d, sizeof(float)*Csig_sz);
    hipMalloc((void**) &OW_new_d, sizeof(float)*OW_new_sz);
    hipMalloc((void**) &update_weight_d, sizeof(float)*update_sz);
    hipMalloc((void**) &OW_t_d, sizeof(float)*OW_t_sz);
    hipMalloc((void**) &B_new_d, sizeof(float)*B_new_sz);
    hipMalloc((void**) &B_temp_d, sizeof(float)*B_temp_sz);
    /*************************************************************************/	

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);
	
    /*************************************************************************/
    //Host to Device
    hipMemcpy(A_d, A_h, sizeof(float)*A_sz, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeof(float)*B_sz, hipMemcpyHostToDevice);
    hipMemcpy(OW_d, OW_h, sizeof(float)*OW_sz, hipMemcpyHostToDevice);
    /*************************************************************************/
    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Feed_forward function --------------------------------------------------
    feed_forward(matArow, matBcol, matBrow, len, OUT_len, OW_d, C_d, Csig_d, OUT_d, A_d, B_d, cuda_ret);


    // Copy device variables from host ----------------------------------------
    //printf("Copying data from device to host..."); fflush(stdout);
    //startTime(&timer);

    /*************************************************************************/
    //Device To Host
    hipMemcpy(C_h, C_d, sizeof(float)*C_sz, hipMemcpyDeviceToHost);
    hipMemcpy(Csig_h, Csig_d, sizeof(float)*Csig_sz, hipMemcpyDeviceToHost);
    hipMemcpy(OUT_h, OUT_d, sizeof(float)*OUT_sz, hipMemcpyDeviceToHost);
    /*************************************************************************/

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	
    //Back_prop_output---------------------------------------------------------
    update_weight_h[0] = (OUT_h[0] - 1.0f) * 0.05f;
    hipMemcpy(update_weight_d, update_weight_h, sizeof(float)*update_sz, hipMemcpyHostToDevice);
    
    printf("\nupdate: %f", update_weight_h[0]);
       
    back_prop_output(OW_d, C_d, OW_new_d, update_weight_d, OW_t_d, cuda_ret);
    
    printf("\nOW_orig: %f", OW_h[1]);
	
    back_prop_hidden(B_d, update_weight_d, A_d, OW_d, OW_t_d, B_new_d, B_temp_d, cuda_ret);
 /*   
    for(int i = 0; i < 100; i++) {
	feed_forward(matArow, matBcol, matBrow, len, OUT_len, OW_new_d, C_d, Csig_d, OUT_d, B_new_d, B_d, cuda_ret);
    	update_weight_h[0] = (OUT_h[0] - 1.0f) * 0.05f;
 	back_prop_output(OW_new_d, C_d, //, update_weight_d, OW_t_d, cuda_ret);
	back_prop_hidden(B_d, update_weight_d, A_d, OW_new_d, OW_t_d, B_new_d, B_temp_d, cuda_ret);
    }
*/
    hipMemcpy(OW_new_h, OW_new_d, sizeof(float)*OW_new_sz, hipMemcpyDeviceToHost);
    hipMemcpy(OW_h, OW_d, sizeof(float)*OW_sz, hipMemcpyDeviceToHost);
    hipMemcpy(OW_t_h, OW_t_d, sizeof(float)*OW_t_sz, hipMemcpyDeviceToHost);
    hipMemcpy(B_new_h, B_new_d, sizeof(float)*B_new_sz, hipMemcpyDeviceToHost);
    hipMemcpy(update_weight_h, update_weight_d, sizeof(float)*update_sz, hipMemcpyDeviceToHost);	
    hipMemcpy(B_h, B_d, sizeof(float)*B_sz, hipMemcpyDeviceToHost);
    hipMemcpy(B_temp_h, B_temp_d, sizeof(float)*B_temp_sz, hipMemcpyDeviceToHost);
    hipMemcpy(C_h, C_d, sizeof(float)*C_sz, hipMemcpyDeviceToHost);
	
    printf("\nC: %f/%f/%f/%f/%f", C_h[0],C_h[1],C_h[2],C_h[3],C_h[4]);
    printf("\nOW_new: %f/%f/%f/%f/%f", OW_new_h[0],OW_new_h[1],OW_new_h[2],OW_new_h[3],OW_new_h[4]);
    printf("\nOW: %f/%f/%f/%f/%f ", OW_h[0],OW_h[1],OW_h[2],OW_h[3],OW_h[4]);
    for(int i = 0; i < 20; i++) {
	printf("\nA: %f/ B_new: %f ", A_h[i], B_new_h[i]);
    }	
    printf("\nupdate_weight_h: %f/  \n", update_weight_h[0]);
    printf("B: %f/%f/%f/%f/%f \n", B_h[0],B_h[1],B_h[2],B_h[3]);
    printf("B_temp: %f/%f/%f/%f/%f \n", B_temp_h[0],B_temp_h[1],B_temp_h[2],B_temp_h[3]);
    // Verify correctness -----------------------------------------------------
    printf("Verifying results..."); fflush(stdout);
	std::cout << C_sz;
    verify(A_h, B_h, C_h, OUT_h, matArow, matAcol, matBcol);
    // Free memory ------------------------------------------------------------
    free(A_h);
    free(B_h);
    free(C_h);
    free(OW_h);
    free(OUT_h);
    free(Csig_h);
    free(OW_new_h);
    free(update_weight_h);
    free(B_new_h);
    free(B_temp_h);
    free(OW_t_h);
    /*************************************************************************/
    //CUDA Free
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);    
    hipFree(OW_d);
    hipFree(OUT_d);
    hipFree(Csig_d);
    hipFree(OW_new_d);
    hipFree(update_weight_d);
    hipFree(B_new_d);
    hipFree(B_temp_d);
    hipFree(OW_t_d);
    /*************************************************************************/
    return 0;
}
